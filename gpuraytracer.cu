#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

// 3D vector struct
struct Vec3 {
    float x, y, z;
    __device__ Vec3 operator+(Vec3 b) { return {x+b.x, y+b.y, z+b.z}; }
    __device__ Vec3 operator*(float b) { return {x*b, y*b, z*b}; }
    __device__ float dot(Vec3 b) { return x*b.x + y*b.y + z*b.z; }
    __device__ Vec3 cross(Vec3 b) { 
        return {y*b.z - z*b.y, z*b.x - x*b.z, x*b.y - y*b.x};
    }
    __device__ Vec3 normalized() { 
        float len = sqrtf(x*x + y*y + z*z);
        return {x/len, y/len, z/len};
    }
};

// Sphere object
struct Sphere {
    Vec3 center;
    float radius;
};

// Ray-sphere intersection test (GPU-optimized)
__device__ bool hit_sphere(Sphere s, Vec3 ray_origin, Vec3 ray_dir, float &t) {
    Vec3 oc = ray_origin + s.center * -1.0f;
    float a = ray_dir.dot(ray_dir);
    float b = 2.0f * oc.dot(ray_dir);
    float c = oc.dot(oc) - s.radius*s.radius;
    float discr = b*b - 4*a*c;
    if (discr < 0) return false;
    t = (-b - sqrtf(discr)) / (2.0f*a);
    return true;
}

// CUDA kernel to render scene
__global__ void render_kernel(uchar4 *pixels, int width, int height, Sphere *spheres, int num_spheres) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    Vec3 ray_origin = {0, 0, 0};
    Vec3 ray_dir = {(float)x/width - 0.5f, (float)y/height - 0.5f, 1};
    ray_dir = ray_dir.normalized();

    // Trace rays against all spheres
    float closest_t = INFINITY;
    for (int i = 0; i < num_spheres; i++) {
        float t;
        if (hit_sphere(spheres[i], ray_origin, ray_dir, t) {
            if (t < closest_t) closest_t = t;
        }
    }

    // Shade pixel (red if hit, black otherwise)
    if (closest_t < INFINITY) {
        pixels[y*width + x] = {255, 0, 0, 255};  // RGBA
    } else {
        pixels[y*width + x] = {0, 0, 0, 255};
    }
}

// Main function
int main() {
    // Image setup
    int width = 1024, height = 768;
    uchar4 *pixels;
    hipMallocManaged(&pixels, width*height*sizeof(uchar4));

    // Scene setup (2 spheres)
    Sphere *spheres;
    int num_spheres = 2;
    hipMallocManaged(&spheres, num_spheres*sizeof(Sphere));
    spheres[0] = {{0, 0, 5}, 1.0f};
    spheres[1] = {{0, -100.5f, 5}, 100.0f};

    // Launch CUDA kernel
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + 15)/16, (height + 15)/16);
    render_kernel<<<blocksPerGrid, threadsPerBlock>>>(pixels, width, height, spheres, num_spheres);
    hipDeviceSynchronize();

    // Save image (simplified; use stb_image.h for actual PNG output)
    FILE *f = fopen("output.ppm", "wb");
    fprintf(f, "P3\n%d %d\n255\n", width, height);
    for (int i = 0; i < width*height; i++) {
        fprintf(f, "%d %d %d ", pixels[i].x, pixels[i].y, pixels[i].z);
    }
    fclose(f);

    hipFree(pixels);
    hipFree(spheres);
    return 0;
}
